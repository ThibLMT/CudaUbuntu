#include <iostream>

#include "def_types.h"
#include "def_global_variables.h"
#include "allocate_variables.cuh"
#include "read_micro.cuh"
#include "init_params.cuh"

#define NB_PART 200
#define SYSSIZEX 5
#define SYSSIZEY 5
#define SYSSIZEZ 14
#define UNITY 0.010000
#define BOUNDARYCONT 20


int main() {
    char Nlogfile[50]="logfile";
    char Nmicrofile[50]="init_micro";
    // Initialization of some global variables
    //** Flag variables
    Nparafile=(char*)"params.in";
    int i,j,k,k1;
    int nb_taches;
    int iter,niter,imicro;
    char filename[50];
    discrete_elt *particle;
    geom_struct *geom;

    // Initialization of Ierror
    ierror=EXIT_SUCCESS;


    hipMallocManaged(&geom, sizeof(geom_struct));

    // * Get the number of particles
    geom->nb_part = NB_PART;
    // * Get the number of boundary contacts
    geom->nb_bc = BOUNDARYCONT;

    // Allocate the particle array
    int nb_elements = geom->nb_part + geom->nb_bc + 1;
    hipMallocManaged(&particle,nb_elements * sizeof(discrete_elt));

    int blockSize = 256;
    int numBlocks = (nb_elements + blockSize - 1)/blockSize;

    // Sets all particle members to 0
    initialize_particle<<<numBlocks,blockSize>>>(particle,geom);
    hipDeviceSynchronize();

    microfile_read_particle(Nmicrofile,particle,geom);
    microcontfile_read_contact(Nmicrofile,particle,geom);

    //********************
    // Initialize parameters
    // Bulk parameters
    read_table_mat(prop_mat_part);

    // Friction parameters
    prop_mat_part->mu_gg=0.3; //!< Friction coefficient grain-grain
    prop_mat_part->mu_gw=0.3; //!< Friction coefficient grain-wall
    // -- Rolling resistant parameters
    prop_mat_part->mu_roll_gg=0.01; //!< Rolling friction coefficient grain-grain
    prop_mat_part->mu_roll_gw=0.01;
    // Set bulk forces
    gravity.x=0.0;
    gravity.y=0.0;
    gravity.z=-9.81;  // m.s-2

    //adimention of length
    adi_params(prop_mat_part,geom);

    // Frees allocated memory
    hipFree(particle);
    hipFree(geom);
    return 0;
}

