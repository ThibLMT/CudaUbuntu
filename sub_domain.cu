#include "hip/hip_runtime.h"
//
// Created by ThibLMT on 24/03/2021.
//

#include "sub_domain.cuh"
#include <stdio.h>
#include <math.h>

__global__ void initialize_backgrid(unsigned int *backgrid,int *backgrid_insert,geom_struct *geom)
{
    int size_backgrid,size_backgrid_insert;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    size_backgrid_insert=geom->sizex*geom->sizey*geom->sizez;
    size_backgrid=size_backgrid_insert*geom->sizel;

    for (int i = index; i < size_backgrid_insert; i+= stride)
    {
        backgrid_insert[i] = 0;
    }

    for (int i = index; i < size_backgrid; i+= stride)
    {
        backgrid[i] = 0;
    }
}